#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include<sys/time.h>
#include <hip/hip_cooperative_groups.h>
#define N 200
#define ITER 10

namespace cg = cooperative_groups;
extern "C" __global__ void singleKerneldecompose(float *A, float *pivots)
{
        int blockID = blockIdx.x;
        int threadId = threadIdx.x;
        float p = 0;
	cg::grid_group grid = cg::this_grid();
      	int bid; 
	int j;
	for(int i=1;i<N;i++){
	for(j=0;j<ITER;j++){
		bid = j*(N/ITER) + blockID;
                if(bid >= i){
                        p = A[bid * N + i - 1]/A[(i - 1)*N + i - 1];
                        A[bid*N + threadId] -= p * A[(i-1)*N + threadId];
                        A[bid*N + i-1] = p;
                }	
	 }
	 cg::sync(grid);
	 }
}


void printA(float *A){
        for(int i=0;i<N;i++){
                        for(int j=0;j<N;j++)
                                printf(" %8.2f ", A[i*N + j]);
                printf("\n");
                }

}


int main(int argc, char *argv[]){ 
        float *A;
	float *pivots;
	hipMallocManaged(reinterpret_cast<void **>(&A), sizeof(float) * (N * N));
	hipMallocManaged(reinterpret_cast<void **>(&pivots), sizeof(float) * (N));
	hipEvent_t start, stop;
        float time;

	for(int i=0;i<N*N;i++)
		A[i] = (float)(rand()%100);;
	
	/*for(int i=0;i<N;i++){
		for(int j=0;j<N;j++)
		printf(" %6.2f ", A[i*N + j]);
	printf("\n");
	}*/

	hipEventCreate(&start);
        hipEventCreate(&stop);

	printf("\n\n");
        int sMemSize = sizeof(double) * N;	
	void *kernelArgs[]={(void*)&A, (void*)&pivots};
	hipEventRecord(start, 0);
	hipLaunchCooperativeKernel((void *)singleKerneldecompose,
                                              N/ITER, N, kernelArgs,
                                              sMemSize, NULL);
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();
	hipEventElapsedTime(&time, start, stop);
	//printA(A);
	printf("\n \n GPU kernel execution time = %f ms\n",time);
	
	
}
