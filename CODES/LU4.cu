#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include<sys/time.h>
#include <hip/hip_cooperative_groups.h>
#define N 128
#define ITER 2

__global__ void decompose(float *A, float *pivots, int iteration)
{
	int blockID = blockIdx.x;
	int threadId = threadIdx.x;
	int bid;
	float p = 0;
	int i;
	for(i=0;i<ITER;i++){
		bid = i*(N/ITER) + blockID;
		if(bid >= iteration){
			p = A[bid * N + iteration - 1]/A[(iteration - 1)*N + iteration - 1];
			A[bid*N + threadId] -= p * A[(iteration-1)*N + threadId];
			A[bid*N + iteration-1] = p;
		}
	}
}

void printA(float *A){
	for(int i=0;i<N;i++){
                        for(int j=0;j<N;j++)
                                printf(" %8.2f ", A[i*N + j]);
                printf("\n");
                }

}
int main(int argc, char *argv[]){ 
        float *A;
	float *pivots;
        float *dev_a, *dev_pivots;
	int *devItr;
        A=(float *)malloc(sizeof(float)*N*N);
	hipEvent_t start, stop;
        float time;
        float totalTime=0;
	
	hipMalloc ( (void**)&dev_a, N*N* sizeof (float) );
        hipMalloc ( (void**)&dev_pivots, N*sizeof (float) );
        hipMalloc ( (void**)&devItr, sizeof (int) );

        pivots=(float *)malloc(sizeof(float)*N);
	for(int i=0;i<N*N;i++)
		A[i] = (float)(rand()%100);;

	hipMemcpy(dev_a, A, N*N*sizeof(float), hipMemcpyHostToDevice);
	
	/*for(int i=0;i<N;i++){
		for(int j=0;j<N;j++)
		printf(" %6.2f ", A[i*N + j]);
	printf("\n");
	}*/

	//printf("\n\n");

	for(int i=1;i<N;i++)
		pivots[i] = A[(i)*N]/A[0];

	hipMemcpy(dev_pivots, pivots, N*sizeof(float), hipMemcpyHostToDevice);

	hipEventCreate(&start);
        hipEventCreate(&stop);
	for(int i=1;i<N;i++) {
	hipEventRecord(start, 0);
	decompose<<<N/ITER,N>>>(dev_a,dev_pivots,i);	
	hipEventRecord(stop, 0);
	hipDeviceSynchronize();	
	//printf("\n");
	hipMemcpy(A, dev_a, N*N*sizeof(float), hipMemcpyDeviceToHost);
        hipEventElapsedTime(&time, start, stop);
        totalTime += time;
	}
	//printA(A);	
        printf("\n \n GPU kernel execution time = %f ms\n",totalTime);
	
}
